#include "hip/hip_runtime.h"
#include "../global_conf.h"
#include "bpt-gpustm.h"
#include "bpt-gpustm-kernel-tx.cuh"
#include "bpt-gpustm-kernel-tree.cuh"
#include "bpt-gpustm-kernel-def.h"
#include "bpt-gpustm-kernel-aux.cuh"
#include "bpt-gpustm-kernel-radical-search.cuh"
#include "../cuda_utils.h"

#include "hipcub/hipcub.hpp" 
using namespace std;
using namespace gpu_stm_nsp_35; 


#define Radical_Max_Times 100 
namespace gpu_stm_nsp_35{
__global__ void build_index_up(g_tree_t g_tree,int index_size,int layer){
/*{{{*/
    const int tid=blockDim.x * blockIdx.x + threadIdx.x; // thread id 所有issue的线程的index
    const int warp_id = (blockDim.x * blockIdx.x + threadIdx.x) / WARPSIZE; 
    const int thread_id=threadIdx.x % WARPSIZE;// thread id in a warp

    offset_t leaf_size=*(g_tree.g_leaf_size);
    offset_t base=(leaf_size+ORDER-1)/ORDER;
    offset_t upNum=(base+ORDER-1)/ORDER;
    offset_t last_upnum=base;
    offset_t last_base=0;
    // layer 从0 开始，表示倒数第二层的创建
    for(int i=0;i<layer;i++){
        last_base=base;
        base=base+upNum;
        last_upnum=upNum;
        upNum=(upNum+ORDER-1)/ORDER;
        if(last_upnum==1){
            return;
        }
    }

    if(warp_id>=upNum){
        return ;
    }
    volatile indexArr_t* index_section=&(g_tree.g_index_section[base+warp_id]);
    offset_t lastNodeId=last_base+warp_id*WARPSIZE; //last node id 指双层索引结构中的下标；
      
    key_t key=g_tree.g_index_section[lastNodeId+thread_id].key[0];

    if(lastNodeId+thread_id>=base){
        key=MAX_KEY;
    }
    //    nodeId=lastNodeId+thread_id;
    if(tid>=last_upnum){
        key=MAX_KEY;
       //     nodeId=-1;
    }
    index_section->isLast=0;
    index_section->key[thread_id]=key;
        //index_section->children[thread_id]=nodeId;  使用prefix sum

    g_tree.g_prefix_sum[base+warp_id]=lastNodeId;

    if(thread_id%(ORDER/index_size)==0){
        index_section->index[thread_id/(ORDER/index_size)]=key;
/*        if(key==0){

                printf("~~~~~~~~Inner~~~ thread_id:%d ,layer:%d, key:%d, warpId:%d, lastNodeId:%d, lastMinKey:%d, nodeId:%d\n",thread_id,layer,key,warp_id,lastNodeId,g_tree.g_index_section[lastNodeId+thread_id].key[0],base+warp_id);
        }
*/
    }
    __threadfence();
    if(upNum==1&&warp_id==0&&thread_id==0){
        *(g_tree.g_index_root) = base;
        printf("leaf size: %d, first_leaf_Id: %d, root_id:%d,firstLeafMinKey:%d\n",leaf_size,*(g_tree.g_first_leaf),base,g_tree.g_key_section[*(g_tree.g_first_leaf)].key[0]);
    //    for(int i=0;i<ORDER;i++){
    //        printf("-- key[%d]= %d, \n",i,g_tree.g_index_section[base].key[i]);
    //    }
    }

    return ;

    /*}}}*/
}
__global__ void build_index_kernel(g_tree_t g_tree,int index_size){
/*{{{*/

    const int tid=blockDim.x * blockIdx.x + threadIdx.x; // thread id 所有issue的线程的index
    const int warp_id = (blockDim.x * blockIdx.x + threadIdx.x) / WARPSIZE; 
    const int thread_id=threadIdx.x % WARPSIZE;// thread id in a warp

    offset_t leaf_size=*(g_tree.g_leaf_size);
    offset_t warp_start=warp_id*WARPSIZE;
    if(warp_start>=leaf_size){
        return ;
    }
    key_t key=g_tree.g_leaf_section[warp_start+thread_id].minKey;
    offset_t nodeId=g_tree.g_leaf_section[warp_start+thread_id].nodeId;

    // node 结构：
    //keys      [k,k,k,...,k,MAX]   31个key
    //child     [id,id,id...,id]    32个child

    if(tid>=leaf_size||warp_start+thread_id>=leaf_size){
        key=MAX_KEY;
        nodeId=-1;
    }
   // if(thread_id ==WARPSIZE-1){ //每个node中最后一个key设置为MAX
   //     key=MAX_KEY; // 将最后一个warp中超过leaf size 的值设置为MAX KEY;
   // }
    volatile indexArr_t *index_section=&(g_tree.g_index_section[warp_id]);
    
    index_section->isLast=1; //倒数第二层节点置位1， 此时需要node的child中记录的nodeid为g_key_section中的数组索引
    __threadfence();
    __syncwarp();
    index_section->key[thread_id]=key;
    __threadfence();
    index_section->children[thread_id]=nodeId;  //使用prefix_sum，但是最后一层需要child指针， 只有最后一层使用children去找leaf node
    //  最后一层index 的prefixsum
    
    g_tree.g_prefix_sum[warp_id]=-1; //表示当前是最后一层索引
    __threadfence();
    if(thread_id%(ORDER/index_size)==0){
        index_section->index[thread_id/(ORDER/index_size)]=key;
        if(key==0){

            printf("~~~~~~~~lastInner~~~ thread_id:%d , key:%d, nodeId:%d\n",thread_id,key,warp_id);
        }
    }

    __threadfence();

    return;

/*}}}*/
}

__global__ void collect_leafnodeInfo(g_tree_t g_tree){
/*{{{*/
    offset_t cur_leaf=*(g_tree.g_first_leaf);
    int count=0;
    while(cur_leaf!=-1){
        key_t minKey=g_tree.g_key_section[cur_leaf].key[0];
        g_tree.g_leaf_section[count].minKey=minKey;
        g_tree.g_leaf_section[count].nodeId=cur_leaf;
        cur_leaf=g_tree.g_next_section[cur_leaf].nodeid;
        count++;
    }
    *(g_tree.g_leaf_size)=count;
    __threadfence();
   // printf(".......leaf size: %d, lastNodeMinKey: %d, nodeid:%d ,\n",count,g_tree.g_leaf_section[count-1].minKey,g_tree.g_leaf_section[count-1].nodeId);
    __syncwarp();

/*}}}*/
}
__global__ void query_kernel( g_tree_t g_tree,
        g_data_t g_data,
        int index_size,
        int boundary,       //从第boundary个tx开始
        int size           //算上boundary大小的size (这个size是以mission个数为单位.) 
        ){
    /*{{{*/
    //const int tx_id = ((blockDim.x * blockIdx.x + threadIdx.x) / WARPSIZE) * query_queue_length + boundary;
    
    //const int tx_thread_id = (gridDim.x * blockDim.x + threadIdx.x) % ORDER;
    const int tx_id_in_block = threadIdx.x / WARPSIZE;
    
    //if (tx_id * tx_size_get >= size) return; //把一整个warp都在size外面的扔掉.
    
    //const int id_in_ntg_get = threadIdx.x % ntg_size_get;
    //const int ntg_id_in_block_get = threadIdx.x / ntg_size_get;
    //const int ntg_id_in_tx_get = threadIdx.x % WARPSIZE / ntg_size_get;
    __shared__ int cacheNodeId_ss[Block_Dim_Get/WARPSIZE];
    int &cacheNodeId_s = cacheNodeId_ss[tx_id_in_block];
    cacheNodeId_s = -1;
    __syncwarp();
    bool isLastThread = (threadIdx.x % WARPSIZE == WARPSIZE-1); //判断是不是一个warp里最后一个thread
    const int base = boundary * WARPSIZE + ((blockDim.x * blockIdx.x + threadIdx.x) / WARPSIZE * tx_size_get * query_queue_length) + threadIdx.x % WARPSIZE; 


    key_t last_target_key1 = 0;
    for (int i=0; i<query_queue_length;i++) { 
        

        const int mission_id_get = base + i*tx_size_get;        //this value only worked for ntg=1
        //const int mission_id_get = base + i*WARPSIZE;        //this value only worked for ntg=1
        const int mask_get = __ballot_sync(0xffffffff, mission_id_get<size);
        if (mission_id_get >= size) return;
        if (mission_id_get == size-1) isLastThread = true;
       
        
     

        //mission_t mission_get;
        key_t target_key_get;
        ans_t *ans_get;
        
#ifdef ABORT_COUNT
        //init 
        g_data.g_roll_back_count[mission_id_get].roll_back_times = 0;
        g_data.g_roll_back_count[mission_id_get].traversal_steps = 0;
        g_data.g_roll_back_count[mission_id_get].leaf_traversal_steps = 0;
        __syncwarp();
#endif        
        
        //mission_get = g_data.g_mission[mission_id_get]; 
        int idx = g_data.g_idx[mission_id_get];
        target_key_get = g_data.g_keys[idx];
        ans_get = &(g_data.g_ans[idx]);   
        
          
        offset_t local_root = *(g_tree.g_index_root);
        if (local_root == -1) {
            *ans_get = -1;
            return ;
        }
        //*ans_get=-1;
        
        //printf("there! id:  %d,   %d, %d\n", mission_id_get, target_key_get, last_target_key1);
        //printf("there! id:  %d, l_id  %d,  %d, %d, %d\n", mission_id_get,i, target_key_get, last_target_key1, cacheNodeId_s);
        index_search_with_cache(target_key_get, last_target_key1, *ans_get,local_root, cacheNodeId_s, isLastThread);
        //printf(" here! id:  %d, l_id  %d,  %d, %d, %d\n", mission_id_get,i, target_key_get, last_target_key1, cacheNodeId_s);
        //printf("here!  id:  %d,   %d, %d\n", mission_id_get, target_key_get, last_target_key1);
        //printf("here!   %d, %d\n", target_key_get, last_target_key1);
        //radical_search(target_key_get, *ans_get, local_root); 

    }

    
    return;

/*}}}*/

}

__global__ void insert_kernel(g_tree_t g_tree,g_data_t g_data,g_lock_t g_lock,int boundary,int maxSize){ 
/*{{{*/
// 只在leafnode 插入
#ifdef ONE_KERNEL
    const int size=maxSize;
#else 
    const int size=(boundary*tx_size_get<maxSize)?boundary*tx_size_get:maxSize;
#endif
    const int warp_id =(blockDim.x* blockIdx.x+threadIdx.x) / WARPSIZE;
    const int thread_id= threadIdx.x %WARPSIZE;
    const int warp_id_in_block = threadIdx.x / WARPSIZE;
    if(warp_id*insert_queue_length>=size) return;

    __shared__ int cacheNodeId_ss[Block_Dim_Put / WARPSIZE];
    int &cacheNodeId_s =cacheNodeId_ss[warp_id_in_block];
    cacheNodeId_s=-1;
    __syncwarp();
   // bool isLastThread=(threadIdx.x % WARPSIZE==WARPSIZE - 1); // 1个warp处理同时只处理1个请求，不需要
    offset_t local_root =*(g_tree.g_index_root); // 使用prefix-sum 

    if(local_root==-1){
        printf("unsupported for empty tree!!!, please use tx_insert_kernel() to build the tree!!! \n");
        return;
    }
    
    __shared__ offset_t nodeId_s[Block_Dim_Put/ WARPSIZE];
    offset_t &nodeId=nodeId_s[warp_id_in_block];
    __shared__ offset_t new_nodeId_s[Block_Dim_Put/WARPSIZE];
    offset_t &new_loc=new_nodeId_s[warp_id_in_block];
    new_loc=-1;
    __shared__ char target_thread_s[Block_Dim_Put / WARPSIZE];
    char &target_thread = target_thread_s[warp_id_in_block];
    char selfFlag;
    offset_t old_nodeId;

    const int base=warp_id* insert_queue_length;
    key_t last_target_key;  //for buffer
    for(int l_i=0;l_i<insert_queue_length;l_i++){
        const int mission_id = base+l_i;
        key_t target_key;
        mission_t mission;
        offset_t target_val;
        ans_t * ans;
#ifdef ABORT_COUNT 
        g_data.g_roll_back_count[mission_id].roll_back_times = -1;
        g_data.g_roll_back_count[mission_id].traversal_steps = 0;
        g_data.g_roll_back_count[mission_id].leaf_traversal_steps = 0;
        __syncwarp();
#endif 
        if(mission_id >= size) return;

        int idx = g_data.g_idx[mission_id];
        target_key = g_data.g_keys[idx];
        mission = g_data.g_mission[mission_id];
        target_val = g_data.g_vals[idx];
        ans= &(g_data.g_ans[idx]);

        //key_t key;
        //offset_t val;
        __syncwarp();
        int isLast;
#ifndef NO_BUFFER 
        if(cacheNodeId_s>=0&& mission==INSERT){
            nodeId= cacheNodeId_s;
            __syncwarp();
            //insert这里是一个warp处理一个任务.
            //printf("test:%d:  %d - %d = %d\n",threadIdx.x, target_key, last_target_key, target_key - last_target_key);
#ifdef USING_STEP_THRESHOLD
            if (target_key - last_target_key < BUFFER_THRESHOLD) 
#endif
                goto search_list;
        }
#endif
        nodeId=local_root;
        __syncwarp();
        isLast=g_tree.g_index_section[nodeId].isLast;
        key_t key;
        offset_t val;
        key_t right_min;
        key_t moved_key;
        offset_t moved_val;
        while(isLast==0){
            selfFlag=1;
            key=g_tree.g_index_section[nodeId].key[thread_id];
            __syncwarp();

#ifdef ABORT_COUNT
            if (thread_id == 0)
                g_data.g_roll_back_count[mission_id].traversal_steps++;
#endif 


            if(target_key<key){
                selfFlag=0;   
            }
            __syncwarp();
            for(int off=WARPSIZE/2;off>0;off/=2){
                selfFlag+=__shfl_down_sync(0xFFFFFFFF,selfFlag,off,WARPSIZE);
            }
            __syncwarp();
            if(thread_id==0){
                if(selfFlag==0){
                    nodeId=__ldg(&(g_tree.g_prefix_sum[nodeId]));
                }else{
                    nodeId=__ldg(&(g_tree.g_prefix_sum[nodeId]))+selfFlag-1;
                }
            
            }
            __syncwarp();

            isLast=g_tree.g_index_section[nodeId].isLast;
        }
        
        // 搜索最后一层index    last_inner_node
        
        selfFlag=1;
        key=g_tree.g_index_section[nodeId].key[thread_id];
    
#ifdef ABORT_COUNT
        if (thread_id == 0)
            g_data.g_roll_back_count[mission_id].traversal_steps++;
#endif 

        if(target_key<key){
            selfFlag=0;
        }
        __syncwarp();
        for(int off=WARPSIZE/2;off>0;off/=2){
            selfFlag+=__shfl_down_sync(0xffffffff,selfFlag,off,WARPSIZE);
        }
        __syncwarp();
        if(thread_id==0){
            if(selfFlag==0){
                nodeId=g_tree.g_index_section[nodeId].children[selfFlag];
            }else{
                nodeId=g_tree.g_index_section[nodeId].children[selfFlag-1];
            }
        }
        __syncwarp();
       // continue;
     
search_list:
        old_nodeId=nodeId;

leaf_start:

#ifdef ABORT_COUNT
        if (thread_id == 0) {
            g_data.g_roll_back_count[mission_id].roll_back_times++;
        }
        __threadfence();
#endif  
        nodeId=old_nodeId;
        __threadfence();
       // __syncwarp();
        // 获取leafnode信息之前需要获取锁
        offset_t minKey= MAX_KEY;
       // offset_t nextLeaf=-1;
        key_t nodeInfo;
        minKey=g_tree.g_val_section[nodeId].val[ORDER-1];

#ifdef ABORT_COUNT
        //第一个Leafnode
        if (thread_id == 0)  {

            g_data.g_roll_back_count[mission_id].traversal_steps++;
            g_data.g_roll_back_count[mission_id].leaf_traversal_steps++;
        }
#endif 
        
        while(target_key>=minKey){
            nodeId=g_tree.g_next_section[nodeId].nodeid;
            if(nodeId<0 || nodeId>=MAX_NODE_NUM){
                goto leaf_start;
            }
            nodeInfo=g_tree.g_key_section[nodeId].key[ORDER-1];
            if(getNodeLevel(nodeInfo)!=0){
                goto leaf_start;
            }
            //minKey=MAX_KEY;
            minKey=g_tree.g_val_section[nodeId].val[ORDER-1];

#ifdef ABORT_COUNT
            if (thread_id == 0) { 
                    
                g_data.g_roll_back_count[mission_id].traversal_steps++;
                
                g_data.g_roll_back_count[mission_id].leaf_traversal_steps++;
            }
#endif 
        }

        if(thread_id==0){
            
                version_t new_lock= constructLock(0,true);
                version_t old_lock= constructLock(0,false);
                if(atomicCAS((version_t*)&(g_lock.g_lock_table[nodeId]),old_lock,new_lock)!=old_lock){
                //    printf("lock fail\n");
                    goto leaf_start;
                }
            
        }

        __syncwarp();
        key_t firstKey=g_tree.g_key_section[nodeId].key[0];
        key_t superKey=g_tree.g_val_section[nodeId].val[ORDER-1];
        if((target_key<firstKey&&nodeId!=0) || target_key>= superKey ){
            g_lock.g_lock_table[nodeId]=constructLock(0,false);
            __threadfence();
            goto leaf_start;
        }
        

        __syncwarp();
       // if(thread_id==0)
       //     printf("找到leafnode %d\n",mission_id);
        nodeInfo=g_tree.g_key_section[nodeId].key[ORDER-1];

        key=g_tree.g_key_section[nodeId].key[thread_id];
        val=g_tree.g_val_section[nodeId].val[thread_id];
      
        __syncwarp();
        *ans=-1;
        __syncwarp();
        if(thread_id==ORDER-1){
            key=MAX_KEY;
        } 
        if(key==target_key){
            *ans=val;
            val = target_val;
        }
        __syncwarp();
       
        if(thread_id==ORDER-1){
            key=nodeInfo;
        } 
        __syncwarp();
        if(mission==GET){
            goto next_mission;
        }
        if(__any_sync(0xffffffff,key==target_key)){
            g_tree.g_key_section[nodeId].key[thread_id]=key;
            g_tree.g_val_section[nodeId].val[thread_id]=val;
            goto next_mission;
        }

        //寻找正确的位置
        selfFlag=1;
        if(thread_id==ORDER-1){
            key=MAX_KEY;
        } 
        if(target_key<key){
            selfFlag=0;
        }
        __syncwarp();
        for(int off=WARPSIZE/2;off>0;off/=2){
            selfFlag+=__shfl_down_sync(0xffffffff,selfFlag,off,WARPSIZE);
        }
        __syncwarp();
        if(thread_id==0){
            target_thread=selfFlag;
        }
        __syncwarp();
            
        key_t tmp_key;
        offset_t tmp_val;
        moved_key= __shfl_up_sync(0xffffffff,key,1,ORDER);
        moved_val= __shfl_up_sync(0xffffffff,val,1,ORDER);
        if(thread_id==target_thread){
            tmp_key= target_key;
            tmp_val=target_val;
        }else if(thread_id> target_thread){
            tmp_key=moved_key;
            tmp_val=moved_val;
        }else{
            tmp_key=key;
            tmp_val=val;
        }
        __syncwarp();
        
        right_min=val;
        __syncwarp();
        if(getNodeSize(nodeInfo)==ORDER-1){
            __syncwarp();
            //创建新的leafnode,并将两个node 中的key平均分配，添加next指针
            key_t up_key=__shfl_sync(0xffffffff,tmp_key,ORDER/2,ORDER);
            key=MAX_KEY;
            val=-1;
            if(thread_id<ORDER/2){
                key=tmp_key;
                val=tmp_val;
            }else if(thread_id==ORDER-1){
                key=info(ORDER/2,0);
                val=up_key;
            }
            __syncwarp();
            g_tree.g_key_section[nodeId].key[thread_id]=key;
            g_tree.g_val_section[nodeId].val[thread_id]=val;
            __syncwarp();
           

            //new node 
            key=MAX_KEY;
            val=-1;
            tmp_key=__shfl_down_sync(0xffffffff,tmp_key,ORDER/2,ORDER);
            tmp_val=__shfl_down_sync(0xffffffff,tmp_val,ORDER/2,ORDER);
            if(thread_id<(ORDER/2)){
                key=tmp_key;
                val=tmp_val;
            }else if(thread_id==ORDER-1){
                key=info(ORDER/2,0);
                val=right_min;
            }
            __syncwarp();
        
            if(thread_id==0){
            
                new_loc=atomicAdd((offset_t*)(g_tree.g_tree_size),1);
               // offset_t tmp_loc=new_loc;
                g_lock.g_lock_table[new_loc]=constructLock(0,false);
                offset_t old_next=g_tree.g_next_section[nodeId].nodeid;
                g_tree.g_next_section[nodeId].nodeid=new_loc;
                g_tree.g_next_section[new_loc].nodeid=old_next;
            }
            __syncwarp();
            g_tree.g_key_section[new_loc].key[thread_id]=key;
            g_tree.g_val_section[new_loc].val[thread_id]=val;
            __syncwarp();
           
        }else{
       
            if(thread_id==ORDER-1){
                tmp_key= infoSizeInc(nodeInfo);
                tmp_val=val;
            }
            __syncwarp();
            g_tree.g_key_section[nodeId].key[thread_id]=tmp_key;
            g_tree.g_val_section[nodeId].val[thread_id]=tmp_val;
        }
next_mission:
        if(thread_id==0){
           // version_t old_lock=constructLock(0,true);
          //  version_t new_lock=  
            g_lock.g_lock_table[nodeId]=constructLock(0,false);

          //  atomicCAS((version_t *)&(g_lock.g_lock_table[nodeId]),old_lock,new_lock);
           // printf("释放锁 %d\n",nodeId);
        }
        cacheNodeId_s=nodeId;
        last_target_key = target_key;       //for buffer threshold
        __syncwarp();
    }


/*}}}*/
}

__global__ void tx_insert_kernel(g_tree_t g_tree, 
        g_data_t g_data,
        g_tx_t g_tx,
        int boundary,
        int maxSize
        ) {
/*{{{*/   
#ifdef ONE_KERNEL
    const int size=maxSize;
#else 
    const int size=(boundary*tx_size_get<maxSize)?boundary*tx_size_get:maxSize;
#endif
    const int tx_id = (blockDim.x * blockIdx.x + threadIdx.x) / WARPSIZE;
    const int tx_thread_id = threadIdx.x % WARPSIZE;
    const int tx_id_in_block = threadIdx.x / WARPSIZE;
    if (tx_id * tx_size_get >= size) return; //把一整个warp都在size外面的扔掉.


    //每WARPSIZE个THREAD共用一个wrtset,  一个wrtset最多装wrtSetSize个记录， 包括location, version, key_arr, val_arr。
    
    //every ORDER threads use one rdset_s and wrtset_s 
    __shared__ rdset_t rdset_ss[Block_Dim_Put/WARPSIZE];   
    rdset_t &rdset_s = rdset_ss[tx_id_in_block];
    __shared__ wrtset_t wrtset_ss[Block_Dim_Put/WARPSIZE];
    wrtset_t &wrtset_s = wrtset_ss[tx_id_in_block];
    
    wrtset_content_t wrtset;

    
S_ROOT: 
    
    //ROOT相关
    offset_t local_root = *(g_tree.g_root);
    version_t root_ver;

    if (local_root == -1) {
        // 正经的判断并处理new tree的情况.
        if (tx_read_root()==false) goto S_ROOT;
        __syncwarp();
        if (local_root != -1) goto S_WORK;
        
        rdset_s.loc[0] = -1;
        rdset_s.loc[1] = -1;
        rdset_s.offset = 0;
        wrtset_s.size = 0;
        
        if (create_new_tree() == false) goto S_ROOT;
        if (tx_commit(true, NULL) == false) goto S_ROOT;
        goto S_ROOT;
    }

S_WORK:
//=============================================================================== 
    
    const int loop_times = tx_size_get / 1;
    for (int l_i=0; l_i<loop_times;l_i++) {
       
        
        const int mission_id = tx_id * tx_size_get + l_i; 
        key_t target_key;       //每个线程保有自己处理的任务
        mission_t mission;      //每个线程保有自己处理的任务
        offset_t target_val;     //每个线程保有自己处理的任务
        ans_t * ans;            //每个线程保有自己处理的任务
       
        if (mission_id >= size) return;
        
        
        int idx = g_data.g_idx[mission_id];
        target_key = g_data.g_keys[idx];
        mission = g_data.g_mission[mission_id];
        target_val = g_data.g_vals[idx];
        ans = &(g_data.g_ans[idx]);   

#ifdef ABORT_COUNT 

        g_data.g_roll_back_count[mission_id].roll_back_times = 0;
        g_data.g_roll_back_count[mission_id].traversal_steps = 0;
        __syncwarp();
#endif 

        
        //每个thread拥有自己负责的那个任务的相关的值
        offset_t tmpCurNodeId;     
        key_t nodeInfo;                      
        key_t key;    
        offset_t val;
        static __shared__ offset_t searched_value_ss[Block_Dim_Put/WARPSIZE];    
        static __shared__ char target_thread_ss[Block_Dim_Put/WARPSIZE];          

        offset_t &searched_value_s = searched_value_ss[tx_id_in_block];         
        char &target_thread_s = target_thread_ss[tx_id_in_block];

        bool backFlag = false;
        
        //record last layer 
        offset_t lastNodeId;
        key_t last_node_info;
        key_t last_key;       //记录 last_key 
        offset_t last_val;    //记录 last_val
        char last_target_thread; 
        

 
TX_START:
#ifndef ENABLE_LL_RB
TX_START1:
#endif 

        //这个函数要对 tmpCurNodeId, nodeInfo, key, val, target_thread_s 进行更新。
        //last系列的不需要。如果node是满的会直接返回false.
        offset_t local_root1 = *(g_tree.g_root);
        wrtset_s.size = 0;

        
        if (radical_search_for_insert(target_key, local_root1)==true) {
            goto S_WRITE; 
        }

        __syncwarp();

 
        //默认两个set内容为空
        rdset_s.loc[0] = -1;
        rdset_s.loc[1] = -1;
        rdset_s.offset = 0;
        wrtset_s.size = 0;
        __syncwarp();

        if (tx_read_root()==false) { goto  TX_START; }
        //到这里 Local_root不可能为-1 

  


        // 初始化searched_value
        searched_value_s = local_root;
        
        __syncwarp();

        //初始化tmpCurNodeId
        tmpCurNodeId = -1;
        __syncwarp();

        
        //search
        do {
/*{{{*/
#ifdef ENABLE_LL_RB
/*{{{*/
            goto BACK;  //这里好难受....有空改成func

TX_START1:
            if (lastNodeId == -1) goto TX_START;

            //默认两个set内容为空
            //rdset_s.loc[0] = -1;
            //rdset_s.loc[1] = -1;
            //rdset_s.offset = 0;
            rdset_s.loc[!rdset_s.offset] = -1; 
            //事实上，我们需要保证我们再重新读last node时，它跟我们第一次读的时候version相同，
            //所以，把那个记录依旧留在rdset中，tx_read就会自动检查它,
            //在这里我们只重置另一个记录
            wrtset_s.size = 0;
            searched_value_s = lastNodeId;
            __syncwarp();
            tmpCurNodeId = -1;
            backFlag = true;
            //从这里出来的，searched_value_s所保留的nodeId一定不是leaf的.
            //所以这个循环算上这次至少要跑2次.
            //那么错误的lastNodeId最多存活在下一个循环里。（再下一个就被覆盖掉了.）
BACK: 
/*}}}*/
#endif
            lastNodeId = tmpCurNodeId;
            last_node_info = nodeInfo; 
            last_key = key;
            last_val = val;
            last_target_thread = target_thread_s;
           
            tmpCurNodeId = searched_value_s;

            // read node
            if (read_node(tmpCurNodeId, key, val, nodeInfo)==false) { 
                goto TX_START1;
            }
            
            __syncwarp();
            
 //=========提前分裂 situation one 提前分裂满的internal node
/*{{{*/
            if ((getNodeSize(nodeInfo)==ORDER-1)&&(!isLeaf(nodeInfo))) {
                
                if (backFlag) goto TX_START;
                int up_key = -1;
                //Internal node split 提前分裂
                if (node_splitting(tmpCurNodeId, key, val, nodeInfo, up_key)==false) goto TX_START1;
                char place_for_new_loc;
                if (lastNodeId == -1){
                    if (insert_new_root(up_key, tmpCurNodeId, getNodeLevel(nodeInfo)+1, place_for_new_loc)==false) goto TX_START;
                    if (tx_commit(true, place_for_new_loc)==false) goto TX_START;
                }
                else {
                    if (insert_node(lastNodeId, up_key, last_key, last_val, last_target_thread, last_node_info, place_for_new_loc)==false) goto TX_START1;
                    if (tx_commit(false, place_for_new_loc)==false) goto TX_START1;
                }
                //tx_commit
                goto TX_START;
            }
            __syncwarp();
/*}}}*/
            //get target 
            get_target(key, val, target_key, searched_value_s, target_thread_s ); 
            
            __syncwarp();
            
            backFlag = false;
            
               
            /*}}}*/
        }while (!isLeaf(nodeInfo)); 
       //==================================SEARCH DONE==================================== 

S_WRITE:

        __syncwarp();
     

        *ans = -1;
     
        __syncwarp();
        //判断是否找到结果。
        //如果能找到结果，结果必然是在target_thread_s-1的位置。
        if (key == target_key) { 
            *ans = val;
        }
        //有结果的赋值为结果，没有结果的一律-1
       
        //test
       
        __syncwarp();
       
        if (mission == GET) {       //理论上这么应该不会再触发了
            continue;
        }

        __syncwarp();
        
        int up_key = -1; 

        if (insert_leaf(tmpCurNodeId, nodeInfo, target_key, target_val, key, val, target_thread_s, up_key) == false) {
            goto TX_START1;
        }
        if (up_key!=-1) {
            /*if (backFlag)  {
                printf("noway!\n");
                goto TX_START;
            }*/
            char place_for_new_loc;
            if (lastNodeId == -1) {
                if (insert_new_root(up_key, tmpCurNodeId,getNodeLevel(nodeInfo)+1, place_for_new_loc) == false) goto TX_START;
                if (tx_commit(true, place_for_new_loc)==false) goto TX_START;
            }
            else {
                if (insert_node(lastNodeId, up_key, last_key, last_val, last_target_thread, last_node_info, place_for_new_loc)==false) goto TX_START1; 
                if (tx_commit(false, place_for_new_loc)==false) goto TX_START1;
            }
        }
        else {
            if (tx_commit(false, NULL)==false) goto TX_START1;
        }
        
        //if (tx_thread_id==0) printf("success return %d\n", target_key);
        __syncwarp();
        continue; 





    }
    
 
    

    return;
/*}}}*/
}



void GPU_STM_Tree::launchBuildIndexKernel(){
    
    dim3 grid_dim1(1);
    dim3 block_dim1(1);
    collect_leafnodeInfo<<<grid_dim1, block_dim1>>>(g_tree);
    cout<< "build index... Middle!"<<endl;
    dim3 grid_dim2(Grid_Dim_prefix*2);
    dim3 block_dim2(Block_Dim_Get);
    build_index_kernel<<<grid_dim2, block_dim2>>>(g_tree,index_size);

    for(int i=0;i<6;i++){
        build_index_up<<<grid_dim2,block_dim2>>>(g_tree,index_size,i);
    }
    
    cout<< "build index...Done! total threads Num:"<<Grid_Dim*Block_Dim_Get<<endl;
    hipError_t error=hipGetLastError();
    printf("cuda error: %s\n",hipGetErrorString(error));

}





void GPU_STM_Tree::launchKernel(int size, bool special, Time_Measure &t) {
#ifdef ONE_KERNEL
    int grid_dim1 = (h_device_working_num2 + (Block_Dim_Put-1)) / Block_Dim_Put;
    int grid_dim2=0;
#else 
    int grid_dim1 = (h_boundary * WARPSIZE + (Block_Dim_Put-1)) / Block_Dim_Put;
    int grid_dim2 = (((h_device_working_num2 - h_boundary * tx_size_get) * ntg_size_get / query_queue_length  + 1)  + (Block_Dim_Get-1)) / Block_Dim_Get;

#endif
      //h_boundary * tx_size_get  insert处理了多少个任务.
    
    //cout<<"grid_dim1: "<<grid_dim1<<endl;
    //cout<<"grid_dim2: "<<grid_dim2<<endl;
    /////cout<<"working size:      "<<h_device_working_num2<<endl;
    //cout<<"boundary:          "<<h_boundary*tx_size_get<<endl;
    t.gpuTimeStart();
    if (grid_dim1 != 0){
        if(insert_count<2){
            //cout<<"tx_insert_kernel"<<endl;
            tx_insert_kernel<<<grid_dim1, Block_Dim_Put>>>(g_tree, g_data, g_tx, h_boundary, h_device_working_num2);
            //hipError_t error=hipGetLastError();
            //printf("cuda error in after tx_insert_kernel: %s\n",hipGetErrorString(error));
        }else{
            //cout<<"insert_kernel"<<endl;
            insert_kernel<<<grid_dim1*(tx_size_get/insert_queue_length),Block_Dim_Put>>>(g_tree,g_data,g_lock,h_boundary,h_device_working_num2);
            //hipError_t error=hipGetLastError();
            //printf("cuda error in after insert_kernel: %s\n",hipGetErrorString(error));
        }
    }
    t.gpuTimeEnd();
//    cout<<"insert done"<<endl;

    t.gpuTimeStart();
    if (grid_dim2 !=0){
        query_kernel<<<grid_dim2, Block_Dim_Get>>> (g_tree, g_data,index_size, h_boundary, h_device_working_num2);
        //search_kernel<<<grid_dim2, Block_Dim_Get>>> (g_tree, g_data, h_boundary, h_device_working_num2);

    }
    t.gpuTimeEnd();
}


void GPU_STM_Tree::launchKernelBefore(int size,Time_Measure &t) {
    t.gpuTimeStart();
    
    //hipcub::DeviceRadixSort::SortPairs<key_t, int>(g_data_phase_1.d_temp_storage, g_data_phase_1.temp_storage_bytes, g_data_phase_1.g_keys, g_data_phase_2.g_keys, g_data_phase_1.g_idx, g_data_phase_2.g_idx, batch_size, 0,32, 0, true);
    hipcub::DeviceRadixSort::SortPairs<key_t, int>(g_data_phase_1.d_temp_storage, g_data_phase_1.temp_storage_bytes, g_data_phase_1.g_keys, g_data_phase_2.g_keys, g_data_phase_1.g_idx, g_data_phase_2.g_idx, size);
    t.gpuTimeEnd();
    t.gpuTimeStart();
    hipcub::DeviceRunLengthEncode::Encode(g_data_phase_2.d_temp_storage, g_data_phase_2.temp_storage_bytes, g_data_phase_2.g_keys, g_data_phase_3.g_keys, g_data_phase_3.g_count, g_data_phase_3.g_num, size);
    
    t.gpuTimeEnd();
    t.gpuTimeStart();
    //scan真正需要的item_num应该是encode得到的g_num,所以要把g_num传回来，然后就能把g_num的值作为scan的参数传入了.
    CUDA_ERROR_HANDLER(hipMemcpy(&h_device_working_num2, g_data_phase_3.g_num, sizeof(int), hipMemcpyDeviceToHost));
    
    hipcub::DeviceScan::ExclusiveSum(g_data_phase_3.d_temp_storage, g_data_phase_3.temp_storage_bytes, g_data_phase_3.g_count, g_data_phase_3.g_idx, h_device_working_num2);
    CUDA_ERROR_HANDLER(hipDeviceSynchronize());

    t.gpuTimeEnd();
    t.gpuTimeStart();
    //int tmpBlockDim = (Block_Dim_Get*4<=512)?Block_Dim_Get*4:512 ;   //4是随便写的
    int tmpBlockDim = 64;
    //dim3 grid_dim( (size + tmpBlockDim-1) / tmpBlockDim );
    dim3 grid_dim( (h_device_working_num2 + tmpBlockDim-1) / tmpBlockDim );
    dim3 block_dim(tmpBlockDim);
    mergeMission<<<grid_dim, block_dim>>>(g_data_phase_2, g_data_phase_3, g_data_phase_4, g_data);
    t.gpuTimeEnd();

    t.gpuTimeStart();
    hipcub::DeviceRadixSort::SortPairs<short, int>(g_data_phase_4.d_temp_storage, g_data_phase_4.temp_storage_bytes, g_data_phase_4.g_mission, g_data.g_mission, g_data_phase_4.g_idx, g_data.g_idx, h_device_working_num2, 0, 2);
    
    t.gpuTimeEnd();
    t.gpuTimeStart();
    //CUDA_ERROR_HANDLER(hipDeviceSynchronize());
   
    dim3 grid_dim2( (h_device_working_num2 + tmpBlockDim-1) / tmpBlockDim );
    getBoundary0<<<1, 1>>>(g_data.g_boundary);
    getBoundary1<<<grid_dim2, block_dim>>>(g_data.g_mission, g_data.g_boundary, h_device_working_num2);
    getBoundary2<<<1, 1>>>(g_data.g_mission, g_data.g_boundary, h_device_working_num2);
    
    CUDA_ERROR_HANDLER(hipMemcpy( &h_boundary, g_data.g_boundary, sizeof(int), hipMemcpyDeviceToHost ));
    CUDA_ERROR_HANDLER(hipDeviceSynchronize());
    t.gpuTimeEnd();
}

void GPU_STM_Tree::launchKernelAfter(int size){
    dim3 grid_dim(Grid_Dim);
    dim3 block_dim(Block_Dim_Get / WARPSIZE * tx_size_get * query_queue_length);
    writebackAns<<<grid_dim, block_dim>>>(g_data_phase_3, g_data, size);
}

__global__ void preprocessKernel(g_tree_t g_tree, g_tx_t g_tx,g_lock_t g_lock){
    
    *(g_tree.g_root) = -1;
    *(g_tree.g_tree_size) = 0;

    *(g_tree.g_first_leaf) = -1;
    *(g_tree.g_leaf_size)=0;
    *(g_tree.g_index_root) = -1;
   
    g_tx.g_lock_table[MAX_NODE_NUM-1] = 0;    //root lock 初始化
    for(int i=0;i<MAX_NODE_NUM;i++){
        g_tree.g_next_section[i].nodeid=-1;
        g_lock.g_lock_table[i]=0; //所有node lock初始化
    }
    
}
__global__ void preprocessKernel_idx_init(int *g_idx_bef) {
    g_idx_bef[blockDim.x * blockIdx.x + threadIdx.x] = blockDim.x * blockIdx.x + threadIdx.x;
}




void GPU_STM_Tree::preprocess() {
    dim3 grid_dim(1);
    dim3 block_dim(1);
    preprocessKernel<<<grid_dim, block_dim>>>(g_tree, g_tx,g_lock);
    dim3 grid_dim2(Grid_Dim);
    dim3 block_dim2(Block_Dim_Get / WARPSIZE * tx_size_get * query_queue_length);
    preprocessKernel_idx_init<<<grid_dim2, block_dim2>>>(g_data_phase_1.g_idx);
    
    //为了得到temp_storage_bytes大小
    hipcub::DeviceRadixSort::SortPairs<key_t, int>(g_data_phase_1.d_temp_storage, g_data_phase_1.temp_storage_bytes, g_data_phase_1.g_keys, g_data_phase_2.g_keys, g_data_phase_1.g_idx, g_data_phase_2.g_idx, batch_size);
    CUDA_ERROR_HANDLER(hipMalloc(&(g_data_phase_1.d_temp_storage), g_data_phase_1.temp_storage_bytes ));

    hipcub::DeviceRunLengthEncode::Encode(g_data_phase_2.d_temp_storage, g_data_phase_2.temp_storage_bytes, g_data_phase_2.g_keys, g_data_phase_3.g_keys, g_data_phase_3.g_count, g_data_phase_3.g_num, batch_size);
    CUDA_ERROR_HANDLER(hipMalloc(&(g_data_phase_2.d_temp_storage), g_data_phase_2.temp_storage_bytes ));

    hipcub::DeviceScan::ExclusiveSum(g_data_phase_3.d_temp_storage, g_data_phase_3.temp_storage_bytes, g_data_phase_3.g_count, g_data_phase_3.g_idx, batch_size);
    CUDA_ERROR_HANDLER(hipMalloc(&(g_data_phase_3.d_temp_storage), g_data_phase_3.temp_storage_bytes ));

    hipcub::DeviceRadixSort::SortPairs<short, int>(g_data_phase_4.d_temp_storage, g_data_phase_4.temp_storage_bytes, g_data_phase_4.g_mission, g_data.g_mission, g_data_phase_4.g_idx, g_data.g_idx, batch_size, 0, 2);
    CUDA_ERROR_HANDLER(hipMalloc(&(g_data_phase_4.d_temp_storage), g_data_phase_4.temp_storage_bytes ));
    //for test
#if 0 
    cout<<"g_tree: "<<endl;
    cout<<(offset_t*)g_tree.g_root<<endl;
    cout<<(offset_t*)g_tree.g_tree_size<<endl;
    cout<<(keyArr_t*)g_tree.g_key_section<<endl;
    cout<<(valArr_t*)g_tree.g_val_section<<endl;
    cout<<"g_data: "<<endl;
    cout<<g_data.g_keys<<endl;
    cout<<g_data.g_vals<<endl;
    cout<<g_data.g_mission<<endl;
    cout<<g_data.g_ans<<endl;
    cout<<"g_tx: "<<endl;
    cout<<g_tx.g_wrtSet_location<<endl;
    cout<<g_tx.g_wrtSet_version<<endl;
    cout<<g_tx.g_wrtSet_content_key<<endl;
    cout<<g_tx.g_wrtSet_content_val<<endl;
    cout<<(version_t*)g_tx.g_lock_table<<endl;
#endif


}


















}
