#include "hip/hip_runtime.h"
#include "../global_conf.h"
#include "bpt-gpustm.h"
#include "bpt-gpustm-kernel-tx.cuh"
#include "bpt-gpustm-kernel-tree.cuh"
#include "bpt-gpustm-kernel-def.h"
#include "bpt-gpustm-kernel-aux.cuh"
#include "../cuda_utils.h"

#include "hipcub/hipcub.hpp" 
using namespace std;
using namespace gpu_stm_nsp_38; 


namespace gpu_stm_nsp_38{
__global__ void tx_insert_kernel(g_tree_t g_tree, 
        g_data_t g_data,
        g_tx_t g_tx,
        int size
        ) {

    const int tx_id = (blockDim.x * blockIdx.x + threadIdx.x) / WARPSIZE;
    const int tx_thread_id = threadIdx.x % WARPSIZE;
    const int tx_id_in_block = threadIdx.x / WARPSIZE;
    if (tx_id * insert_queue_length >= size) return; //把一整个warp都在size外面的扔掉.


    //每WARPSIZE个THREAD共用一个wrtset,  一个wrtset最多装wrtSetSize个记录， 包括location, version, key_arr, val_arr。
    
    //every ORDER threads use one rdset_s and wrtset_s 
    __shared__ rdset_t rdset_ss[Block_Dim/WARPSIZE];   
    rdset_t &rdset_s = rdset_ss[tx_id_in_block];
    __shared__ wrtset_t wrtset_ss[Block_Dim/WARPSIZE];
    wrtset_t &wrtset_s = wrtset_ss[tx_id_in_block];
    
    wrtset_content_t wrtset;
    
    offset_t local_root;
    version_t root_ver;

    
//=============================================================================== 
    
    for (int l_i=0; l_i<insert_queue_length;l_i++) {

        
        key_t target_key;       //每个线程保有自己处理的任务
        mission_t mission;      //每个线程保有自己处理的任务
        offset_t target_val;     //每个线程保有自己处理的任务
        ans_t * ans;            //每个线程保有自己处理的任务
        const int mission_id = tx_id * insert_queue_length + l_i; 
       
        if (mission_id >= size) return;
        
        
        target_key = g_data.g_keys[mission_id];
        mission = g_data.g_mission[mission_id];
        target_val = g_data.g_vals[mission_id];
        ans = &(g_data.g_ans[mission_id]);   

#ifdef ABORT_COUNT 
        g_data.g_roll_back_count[mission_id].roll_back_times = -1;
        g_data.g_roll_back_count[mission_id].traversal_steps = 0;
        g_data.g_roll_back_count[mission_id].leaf_traversal_steps = 0;
        __syncwarp();
#endif 

        
        //每个thread拥有自己负责的那个任务的相关的值
        offset_t tmpCurNodeId;     
        key_t nodeInfo;                      
        key_t key;    
        offset_t val;
        static __shared__ offset_t searched_value_ss[Block_Dim/WARPSIZE];    
        static __shared__ char target_thread_ss[Block_Dim/WARPSIZE];          

        offset_t &searched_value_s = searched_value_ss[tx_id_in_block];         
        char &target_thread_s = target_thread_ss[tx_id_in_block];

        
        //record last layer 
        offset_t lastNodeId ;
        key_t last_node_info;
        key_t last_key;       //记录 last_key 
        offset_t last_val;    //记录 last_val
        char last_target_thread; 
        

 
TX_START:



#ifdef ABORT_COUNT
        __syncwarp();
        if (tx_thread_id == 0)
            g_data.g_roll_back_count[mission_id].roll_back_times++;
        __syncwarp();
#endif  



        __syncwarp();

 

        //默认两个set内容为空
        rdset_s.loc[0] = -1;
        rdset_s.loc[1] = -1;
        rdset_s.offset = 0;
        wrtset_s.size = 0;
        __syncwarp();

        if (tx_read_root()==false) { goto  TX_START; }

        if (local_root == -1) {
            
            if (create_new_tree() == false) goto TX_START;
            if (tx_commit(true, NULL) == false) goto TX_START;
            goto TX_START;
        }
  
        //__syncwarp();
        //if (tx_thread_id == 0){
        //    printf("root: %d, mission_id, %d\n", local_root, mission_id);
        //}
        __syncwarp();


        // 初始化searched_value
        searched_value_s = local_root;
        
        //初始化tmpCurNodeId
        tmpCurNodeId = -1;
        __syncwarp();

        
        //search
        do {
/*{{{*/

            __syncwarp();
            lastNodeId = tmpCurNodeId;
            last_node_info = nodeInfo; 
            last_key = key;
            last_val = val;
            last_target_thread = target_thread_s;
           
            tmpCurNodeId = searched_value_s;
            __syncwarp();

#ifdef ABORT_COUNT 
            if (tx_thread_id == 0)
                g_data.g_roll_back_count[mission_id].traversal_steps++; 
#endif 
            // read node
            if (read_node(tmpCurNodeId, key, val, nodeInfo)==false) { 
                goto TX_START;
            }
            
            __syncwarp();
            
 //=========提前分裂 situation one 提前分裂满的internal node
/*{{{*/
            if ((getNodeSize(nodeInfo)==ORDER-1)&&(!isLeaf(nodeInfo))) {
                
                int up_key = -1;
                //Internal node split 提前分裂
                if (node_splitting(tmpCurNodeId, key, val, nodeInfo, up_key)==false) goto TX_START;
                __syncwarp();
                char place_for_new_loc;
                if (lastNodeId == -1){
                    if (insert_new_root(up_key, tmpCurNodeId, getNodeLevel(nodeInfo)+1, place_for_new_loc)==false) goto TX_START;
                    if (tx_commit(true, place_for_new_loc)==false) goto TX_START;
                }
                else {
                    if (insert_node(lastNodeId, up_key, last_key, last_val, last_target_thread, last_node_info, place_for_new_loc)==false) goto TX_START;
                    if (tx_commit(false, place_for_new_loc)==false) goto TX_START;
                }
                //tx_commit
                goto TX_START;
            }
            __syncwarp();
/*}}}*/
            //get target 
            get_target(key, val, target_key, searched_value_s, target_thread_s ); 
            
            __syncwarp();
            
            
               
            /*}}}*/
        }while (!isLeaf(nodeInfo)); 
       //==================================SEARCH DONE==================================== 


        __syncwarp();
     

        *ans = -1;
     
        __syncwarp();
        //判断是否找到结果。
        //如果能找到结果，结果必然是在target_thread_s-1的位置。
        if (key == target_key) { 
            *ans = val;
        }
        //有结果的赋值为结果，没有结果的一律-1
       
        //test
       
        __syncwarp();
       
        if (mission == GET) {
            continue;
        }

        __syncwarp();
        
        int up_key = -1; 

        if (insert_leaf(tmpCurNodeId, nodeInfo, target_key, target_val, key, val, target_thread_s, up_key) == false) {
            goto TX_START;
        }
        __syncwarp();
        if (up_key!=-1) {
            char place_for_new_loc;
            if (lastNodeId == -1) {
                if (insert_new_root(up_key, tmpCurNodeId,getNodeLevel(nodeInfo)+1, place_for_new_loc) == false) goto TX_START;
                if (tx_commit(true, place_for_new_loc)==false) goto TX_START;
            }
            else {
                if (insert_node(lastNodeId, up_key, last_key, last_val, last_target_thread, last_node_info, place_for_new_loc)==false) goto TX_START; 
                if (tx_commit(false, place_for_new_loc)==false) goto TX_START;
            }
        }
        else {
            if (tx_commit(false, NULL)==false) goto TX_START;
        }
        
        //if (tx_thread_id==0) printf("success return %d\n", target_key);
        __syncwarp();
        
        
        
        continue; 





    }
    
 
    

    return;

}





void GPU_STM_Tree::launchKernel(int size, bool special, Time_Measure &t) {

    //int grid_dim1 = (h_boundary * WARPSIZE + (Block_Dim_Put-1)) / Block_Dim_Put;
    //int grid_dim2 = (h_device_working_num2 - h_boundary * WARPSIZE + (Block_Dim_Get-1)) / Block_Dim_Get;
    
    int requests_per_block = Block_Dim / WARPSIZE * insert_queue_length ; 
    int grid_dim = (h_device_working_num2 + requests_per_block - 1) / requests_per_block;
    //cout<<"grid_dim1: "<<grid_dim1<<endl;
    //cout<<"grid_dim2: "<<grid_dim2<<endl;
    //cout<<"working size:      "<<h_device_working_num2<<endl;
    t.gpuTimeStart();


    if (grid_dim != 0)  {

        tx_insert_kernel<<<grid_dim, Block_Dim>>>(g_tree, g_data, g_tx, h_device_working_num2);
        
       // hipError_t error = hipGetLastError();
       // printf("cuda error: %s\n",hipGetErrorString(error));
    }
    t.gpuTimeEnd();
    
}


void GPU_STM_Tree::launchKernelBefore(int size,Time_Measure &t) {
    t.gpuTimeStart();
    
    //hipcub::DeviceRadixSort::SortPairs<key_t, int>(g_data_phase_1.d_temp_storage, g_data_phase_1.temp_storage_bytes, g_data_phase_1.g_keys, g_data_phase_2.g_keys, g_data_phase_1.g_idx, g_data_phase_2.g_idx, batch_size, 0,32, 0, true);
    hipcub::DeviceRadixSort::SortPairs<key_t, int>(g_data_phase_1.d_temp_storage, g_data_phase_1.temp_storage_bytes, g_data_phase_1.g_keys, g_data_phase_2.g_keys, g_data_phase_1.g_idx, g_data_phase_2.g_idx, size);
    t.gpuTimeEnd();
    t.gpuTimeStart();
    hipcub::DeviceRunLengthEncode::Encode(g_data_phase_2.d_temp_storage, g_data_phase_2.temp_storage_bytes, g_data_phase_2.g_keys, g_data_phase_3.g_keys, g_data_phase_3.g_count, g_data_phase_3.g_num, size);
    
    t.gpuTimeEnd();
    
    
    t.gpuTimeStart();
    //scan真正需要的item_num应该是encode得到的g_num,所以要把g_num传回来，然后就能把g_num的值作为scan的参数传入了.
    CUDA_ERROR_HANDLER(hipMemcpy(&h_device_working_num2, g_data_phase_3.g_num, sizeof(int), hipMemcpyDeviceToHost));
    
    hipcub::DeviceScan::ExclusiveSum(g_data_phase_3.d_temp_storage, g_data_phase_3.temp_storage_bytes, g_data_phase_3.g_count, g_data_phase_3.g_idx, h_device_working_num2);
    CUDA_ERROR_HANDLER(hipDeviceSynchronize());

    t.gpuTimeEnd();
   
    
    
    
    t.gpuTimeStart();
    int tmpBlockDim = 64;
    dim3 grid_dim( (size + tmpBlockDim-1) / tmpBlockDim );
    dim3 block_dim(tmpBlockDim);
    mergeMission<<<grid_dim, block_dim>>>(g_data_phase_2, g_data_phase_3,  g_data);
    t.gpuTimeEnd();
    
    
    //hipError_t error = hipGetLastError();
    //printf("cuda error: %s\n",hipGetErrorString(error));

}

void GPU_STM_Tree::launchKernelAfter(int size){
    dim3 grid_dim((size + Block_Dim - 1) / Block_Dim);
    dim3 block_dim(Block_Dim);
    writebackAns<<<grid_dim, block_dim>>>(g_data_phase_3, g_data, size);
    //hipError_t error = hipGetLastError();
    //printf("cuda error: %s\n",hipGetErrorString(error));
}

__global__ void preprocessKernel(g_tree_t g_tree, g_tx_t g_tx){
    
    *(g_tree.g_root) = -1;
    *(g_tree.g_tree_size) = 0;

    g_tx.g_lock_table[MAX_NODE_NUM-1] = 0;    //root lock 初始化

}
__global__ void preprocessKernel_idx_init(int *g_idx_bef) {
    g_idx_bef[blockDim.x * blockIdx.x + threadIdx.x] = blockDim.x * blockIdx.x + threadIdx.x;
}




void GPU_STM_Tree::preprocess() {
    dim3 grid_dim(1);
    dim3 block_dim(1);
    preprocessKernel<<<grid_dim, block_dim>>>(g_tree, g_tx);
    dim3 grid_dim2((batch_size + Block_Dim - 1)/ Block_Dim);
    dim3 block_dim2(Block_Dim);
    preprocessKernel_idx_init<<<grid_dim2, block_dim2>>>(g_data_phase_1.g_idx);
    
    //为了得到temp_storage_bytes大小
    hipcub::DeviceRadixSort::SortPairs<key_t, int>(g_data_phase_1.d_temp_storage, g_data_phase_1.temp_storage_bytes, g_data_phase_1.g_keys, g_data_phase_2.g_keys, g_data_phase_1.g_idx, g_data_phase_2.g_idx, batch_size);
    CUDA_ERROR_HANDLER(hipMalloc(&(g_data_phase_1.d_temp_storage), g_data_phase_1.temp_storage_bytes ));

    hipcub::DeviceRunLengthEncode::Encode(g_data_phase_2.d_temp_storage, g_data_phase_2.temp_storage_bytes, g_data_phase_2.g_keys, g_data_phase_3.g_keys, g_data_phase_3.g_count, g_data_phase_3.g_num, batch_size);
    CUDA_ERROR_HANDLER(hipMalloc(&(g_data_phase_2.d_temp_storage), g_data_phase_2.temp_storage_bytes ));

    hipcub::DeviceScan::ExclusiveSum(g_data_phase_3.d_temp_storage, g_data_phase_3.temp_storage_bytes, g_data_phase_3.g_count, g_data_phase_3.g_idx, batch_size);
    CUDA_ERROR_HANDLER(hipMalloc(&(g_data_phase_3.d_temp_storage), g_data_phase_3.temp_storage_bytes ));

    //for test
#if 0 
    cout<<"g_tree: "<<endl;
    cout<<(offset_t*)g_tree.g_root<<endl;
    cout<<(offset_t*)g_tree.g_tree_size<<endl;
    cout<<(keyArr_t*)g_tree.g_key_section<<endl;
    cout<<(valArr_t*)g_tree.g_val_section<<endl;
    cout<<"g_data: "<<endl;
    cout<<g_data.g_keys<<endl;
    cout<<g_data.g_vals<<endl;
    cout<<g_data.g_mission<<endl;
    cout<<g_data.g_ans<<endl;
    cout<<"g_tx: "<<endl;
    //cout<<g_tx.g_wrtSet_location<<endl;
    //cout<<g_tx.g_wrtSet_version<<endl;
    //cout<<g_tx.g_wrtSet_content_key<<endl;
    //cout<<g_tx.g_wrtSet_content_val<<endl;
    cout<<(version_t*)g_tx.g_lock_table<<endl;
#endif


}


















}
