#include "hip/hip_runtime.h"
#include "../global_conf.h"
#include "bpt-gpustm.h"
#include "bpt-gpustm-kernel-tx.cuh"
#include "bpt-gpustm-kernel-tree.cuh"
#include "bpt-gpustm-kernel-def.h"
#include "../cuda_utils.h"

#include "hipcub/hipcub.hpp" 
using namespace std;
using namespace gpu_stm_nsp_39; 


namespace gpu_stm_nsp_39{
__global__ void tx_insert_kernel(g_tree_t g_tree, 
        g_data_t g_data,
        g_tx_t g_tx,
        int size
        ) {

    const int tx_id = (blockDim.x * blockIdx.x + threadIdx.x) / WARPSIZE;
    const int tx_thread_id = threadIdx.x % WARPSIZE;
    const int tx_id_in_block = threadIdx.x / WARPSIZE;
    if (tx_id * insert_queue_length >= size) return; //把一整个warp都在size外面的扔掉.


    //每WARPSIZE个THREAD共用一个wrtset,  一个wrtset最多装wrtSetSize个记录， 包括location, version, key_arr, val_arr。
    
    //every ORDER threads use one rdset_s and wrtset_s 
    __shared__ rdset_t rdset_ss[Block_Dim/WARPSIZE];   
    rdset_t &rdset_s = rdset_ss[tx_id_in_block];
    __shared__ wrtset_t wrtset_ss[Block_Dim/WARPSIZE];
    wrtset_t &wrtset_s = wrtset_ss[tx_id_in_block];
    
    wrtset_content_t wrtset;

    version_t root_ver;
    offset_t local_root;

//=============================================================================== 
    
    for (int l_i=0; l_i<insert_queue_length;l_i++) {

        
        key_t target_key;       //每个线程保有自己处理的任务
        mission_t mission;      //每个线程保有自己处理的任务
        offset_t target_val;     //每个线程保有自己处理的任务
        ans_t * ans;            //每个线程保有自己处理的任务
        const int mission_id = tx_id * insert_queue_length + l_i; 
       
        if (mission_id >= size) return;
        
        
        target_key = g_data.g_keys[mission_id];
        mission = g_data.g_mission[mission_id];
        target_val = g_data.g_vals[mission_id];
        ans = &(g_data.g_ans[mission_id]);   

#ifdef ABORT_COUNT 
        g_data.g_roll_back_count[mission_id].roll_back_times = -1;
        g_data.g_roll_back_count[mission_id].traversal_steps = 0;
        g_data.g_roll_back_count[mission_id].leaf_traversal_steps = 0;
        __syncwarp();
#endif 

        
        //每个thread拥有自己负责的那个任务的相关的值
        offset_t tmpCurNodeId;     
        key_t nodeInfo;                      
        key_t key;    
        offset_t val;
        static __shared__ offset_t searched_value_ss[Block_Dim/WARPSIZE];    
        static __shared__ char target_thread_ss[Block_Dim/WARPSIZE];          

        offset_t &searched_value_s = searched_value_ss[tx_id_in_block];         
        char &target_thread_s = target_thread_ss[tx_id_in_block];

        
        //record last layer 
        offset_t lastNodeId;
        key_t last_node_info;
        key_t last_key;       //记录 last_key 
        offset_t last_val;    //记录 last_val
        char last_target_thread; 
        

 
TX_START:



#ifdef ABORT_COUNT
        __syncwarp();
        if (tx_thread_id == 0)
            g_data.g_roll_back_count[mission_id].roll_back_times++;
        __syncwarp();
#endif  



        __syncwarp();

 

        //默认两个set内容为空
        rdset_s.loc[0] = -1;
        rdset_s.loc[1] = -1;
        rdset_s.offset = 0;
        wrtset_s.size = 0;
        __syncwarp();

        if (tx_read_root()==false) { goto  TX_START; }
        
        if (local_root == -1) {
            
            if (create_new_tree() == false) goto TX_START;
            if (tx_commit(true, NULL) == false) goto TX_START;
            goto TX_START;
        }
  
        //__syncwarp();
        //if (tx_thread_id == 0){
        //    printf("root: %d, mission_id, %d\n", local_root, mission_id);
        //}
        __syncwarp();


        // 初始化searched_value
        searched_value_s = local_root;
        
        //初始化tmpCurNodeId
        tmpCurNodeId = -1;
        __syncwarp();

        
        //search
        do {
/*{{{*/

            __syncwarp();
            lastNodeId = tmpCurNodeId;
            last_node_info = nodeInfo; 
            last_key = key;
            last_val = val;
            last_target_thread = target_thread_s;
           
            tmpCurNodeId = searched_value_s;
            __syncwarp();

#ifdef ABORT_COUNT 
            if (tx_thread_id == 0)
                g_data.g_roll_back_count[mission_id].traversal_steps++; 
#endif 
            // read node
            if (read_node(tmpCurNodeId, key, val, nodeInfo)==false) { 
                goto TX_START;
            }
            
            __syncwarp();
            
 //=========提前分裂 situation one 提前分裂满的internal node
/*{{{*/
            if ((getNodeSize(nodeInfo)==ORDER-1)&&(!isLeaf(nodeInfo))) {
                
                int up_key = -1;
                //Internal node split 提前分裂
                if (node_splitting(tmpCurNodeId, key, val, nodeInfo, up_key)==false) goto TX_START;
                __syncwarp();
                char place_for_new_loc;
                if (lastNodeId == -1){
                    if (insert_new_root(up_key, tmpCurNodeId, getNodeLevel(nodeInfo)+1, place_for_new_loc)==false) goto TX_START;
                    if (tx_commit(true, place_for_new_loc)==false) goto TX_START;
                }
                else {
                    if (insert_node(lastNodeId, up_key, last_key, last_val, last_target_thread, last_node_info, place_for_new_loc)==false) goto TX_START;
                    if (tx_commit(false, place_for_new_loc)==false) goto TX_START;
                }
                //tx_commit
                goto TX_START;
            }
            __syncwarp();
/*}}}*/
            //get target 
            get_target(key, val, target_key, searched_value_s, target_thread_s ); 
            
            __syncwarp();
            
            
               
            /*}}}*/
        }while (!isLeaf(nodeInfo)); 
       //==================================SEARCH DONE==================================== 


        __syncwarp();
     

        *ans = -1;
     
        __syncwarp();
        //判断是否找到结果。
        //如果能找到结果，结果必然是在target_thread_s-1的位置。
        if (key == target_key) { 
            *ans = val;
        }
        //有结果的赋值为结果，没有结果的一律-1
       
        //test
       
        __syncwarp();
       
        if (mission == GET) {
            continue;
        }

        __syncwarp();
        
        int up_key = -1; 

        if (insert_leaf(tmpCurNodeId, nodeInfo, target_key, target_val, key, val, target_thread_s, up_key) == false) {
            goto TX_START;
        }
        __syncwarp();
        if (up_key!=-1) {
            char place_for_new_loc;
            if (lastNodeId == -1) {
                if (insert_new_root(up_key, tmpCurNodeId,getNodeLevel(nodeInfo)+1, place_for_new_loc) == false) goto TX_START;
                if (tx_commit(true, place_for_new_loc)==false) goto TX_START;
            }
            else {
                if (insert_node(lastNodeId, up_key, last_key, last_val, last_target_thread, last_node_info, place_for_new_loc)==false) goto TX_START; 
                if (tx_commit(false, place_for_new_loc)==false) goto TX_START;
            }
        }
        else {
            if (tx_commit(false, NULL)==false) goto TX_START;
        }
        
        //if (tx_thread_id==0) printf("success return %d\n", target_key);
        __syncwarp();
        
        
        
        continue; 





    }
    
 
    

    return;

}





void GPU_STM_Tree::launchKernel(int size, bool special, Time_Measure &t) {

    
    int requests_per_block = Block_Dim / WARPSIZE * insert_queue_length ; 
    int grid_dim = (size + requests_per_block - 1) / requests_per_block;
    //cout<<"grid_dim1: "<<grid_dim1<<endl;
    //cout<<"grid_dim2: "<<grid_dim2<<endl;
    //cout<<"working size:      "<<size<<endl;
    t.gpuTimeStart();


    if (grid_dim != 0)  {

        tx_insert_kernel<<<grid_dim, Block_Dim>>>(g_tree, g_data, g_tx, size);
        
       // hipError_t error = hipGetLastError();
       // printf("cuda error: %s\n",hipGetErrorString(error));
    }
    t.gpuTimeEnd();
    
}

__global__ void preprocessKernel(g_tree_t g_tree, g_tx_t g_tx){
    
    *(g_tree.g_root) = -1;
    *(g_tree.g_tree_size) = 0;

    g_tx.g_lock_table[MAX_NODE_NUM-1] = 0;    //root lock 初始化

}





void GPU_STM_Tree::preprocess() {
    dim3 grid_dim(1);
    dim3 block_dim(1);
    preprocessKernel<<<grid_dim, block_dim>>>(g_tree, g_tx);
    
    //for test
#if 0 
    cout<<"g_tree: "<<endl;
    cout<<(offset_t*)g_tree.g_root<<endl;
    cout<<(offset_t*)g_tree.g_tree_size<<endl;
    cout<<(keyArr_t*)g_tree.g_key_section<<endl;
    cout<<(valArr_t*)g_tree.g_val_section<<endl;
    cout<<"g_data: "<<endl;
    cout<<g_data.g_keys<<endl;
    cout<<g_data.g_vals<<endl;
    cout<<g_data.g_mission<<endl;
    cout<<g_data.g_ans<<endl;
    cout<<"g_tx: "<<endl;
    //cout<<g_tx.g_wrtSet_location<<endl;
    //cout<<g_tx.g_wrtSet_version<<endl;
    //cout<<g_tx.g_wrtSet_content_key<<endl;
    //cout<<g_tx.g_wrtSet_content_val<<endl;
    cout<<(version_t*)g_tx.g_lock_table<<endl;
#endif


}


















}
