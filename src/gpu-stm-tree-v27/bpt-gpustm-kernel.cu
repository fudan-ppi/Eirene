#include "hip/hip_runtime.h"
#include "../global_conf.h"
#include "bpt-gpustm.h"
#include "bpt-gpustm-kernel-tx.cuh"
#include "bpt-gpustm-kernel-tree.cuh"
#include "bpt-gpustm-kernel-def.h"
#include "bpt-gpustm-kernel-aux.cuh"
#include "bpt-gpustm-kernel-radical-search.cuh"
#include "../cuda_utils.h"

#include "hipcub/hipcub.hpp" 
using namespace std;
using namespace gpu_stm_nsp_27; 


#define Radical_Max_Times 100 
namespace gpu_stm_nsp_27{
__global__ void tx_insert_kernel(g_tree_t g_tree, 
        g_data_t g_data,
        g_tx_t g_tx,
        int boundary,
        int maxSize
        ) {
    
    const int size = (boundary * tx_size_get < maxSize)?boundary*tx_size_get:maxSize;
    const int tx_id = (blockDim.x * blockIdx.x + threadIdx.x) / WARPSIZE;
    const int tx_thread_id = threadIdx.x % WARPSIZE;
    const int tx_id_in_block = threadIdx.x / WARPSIZE;
    if (tx_id * tx_size_get >= size) return; //把一整个warp都在size外面的扔掉.


    //每WARPSIZE个THREAD共用一个wrtset,  一个wrtset最多装wrtSetSize个记录， 包括location, version, key_arr, val_arr。
    
    //every ORDER threads use one rdset_s and wrtset_s 
    __shared__ rdset_t rdset_ss[Block_Dim_Put/WARPSIZE];   
    rdset_t &rdset_s = rdset_ss[tx_id_in_block];
    __shared__ wrtset_t wrtset_ss[Block_Dim_Put/WARPSIZE];
    wrtset_t &wrtset_s = wrtset_ss[tx_id_in_block];
    
    wrtset_content_t wrtset;

    
S_ROOT: 
    
    //ROOT相关
    offset_t local_root = *(g_tree.g_root);
    version_t root_ver;

    if (local_root == -1) {
        // 正经的判断并处理new tree的情况.
        if (tx_read_root()==false) goto S_ROOT;
        __syncwarp();
        if (local_root != -1) goto S_WORK;
        
        rdset_s.loc[0] = -1;
        rdset_s.loc[1] = -1;
        rdset_s.offset = 0;
        wrtset_s.size = 0;
        
        if (create_new_tree() == false) goto S_ROOT;
        if (tx_commit(true, NULL) == false) goto S_ROOT;
        goto S_ROOT;
    }

S_WORK:
//=============================================================================== 
    
    const int loop_times = tx_size_get / 1;
    for (int l_i=0; l_i<loop_times;l_i++) {
       
        
        const int mission_id = tx_id * tx_size_get + l_i; 
        key_t target_key;       //每个线程保有自己处理的任务
        mission_t mission;      //每个线程保有自己处理的任务
        offset_t target_val;     //每个线程保有自己处理的任务
        ans_t * ans;            //每个线程保有自己处理的任务
       
        if (mission_id >= size) return;
        
        
        int idx = g_data.g_idx[mission_id];
        target_key = g_data.g_keys[idx];
        mission = g_data.g_mission[mission_id];
        target_val = g_data.g_vals[idx];
        ans = &(g_data.g_ans[idx]);   

#ifdef ABORT_COUNT 
        g_data.g_roll_back_count[mission_id].roll_back_times = 0;
        g_data.g_roll_back_count[mission_id].traversal_steps = 0;
        g_data.g_roll_back_count[mission_id].leaf_traversal_steps = 0;
        __syncwarp();
#endif 

        
        //每个thread拥有自己负责的那个任务的相关的值
        offset_t tmpCurNodeId;     
        key_t nodeInfo;                      
        key_t key;    
        offset_t val;
        static __shared__ offset_t searched_value_ss[Block_Dim_Put/WARPSIZE];    
        static __shared__ char target_thread_ss[Block_Dim_Put/WARPSIZE];          

        offset_t &searched_value_s = searched_value_ss[tx_id_in_block];         
        char &target_thread_s = target_thread_ss[tx_id_in_block];

        bool backFlag = false;
        
        //record last layer 
        offset_t lastNodeId;
        key_t last_node_info;
        key_t last_key;       //记录 last_key 
        offset_t last_val;    //记录 last_val
        char last_target_thread; 
        

 
TX_START:
#ifndef ENABLE_LL_RB
TX_START1:
#endif 


#ifdef ABORT_COUNT
        //__syncwarp();
        //if (tx_thread_id == 0)
        //    g_data.g_roll_back_count[mission_id].roll_back_times++;
        //__syncwarp();
#endif  


        //这个函数要对 tmpCurNodeId, nodeInfo, key, val, target_thread_s 进行更新。
        //last系列的不需要。如果node是满的会直接返回false.
        offset_t local_root1 = *(g_tree.g_root);
        wrtset_s.size = 0;
#define THRESHOLD1 1
        int timesss = 0;
        while (timesss < THRESHOLD1) {
            timesss++;
            if (radical_search_for_insert(target_key, local_root1)==true) {

                goto S_WRITE; 
            }
        }

        __syncwarp();
 

        //默认两个set内容为空
        rdset_s.loc[0] = -1;
        rdset_s.loc[1] = -1;
        rdset_s.offset = 0;
        wrtset_s.size = 0;
        __syncwarp();

        if (tx_read_root()==false) { goto  TX_START; }
        //到这里 Local_root不可能为-1 

  


        // 初始化searched_value
        searched_value_s = local_root;
        
        __syncwarp();

        //初始化tmpCurNodeId
        tmpCurNodeId = -1;
        __syncwarp();

        
        //search
        do {
/*{{{*/
#ifdef ENABLE_LL_RB
/*{{{*/
            goto BACK;  //这里好难受....有空改成func

TX_START1:
            if (lastNodeId == -1) goto TX_START;
    
            //默认两个set内容为空
            //rdset_s.loc[0] = -1;
            //rdset_s.loc[1] = -1;
            //rdset_s.offset = 0;
            rdset_s.loc[!rdset_s.offset] = -1; 
            //事实上，我们需要保证我们再重新读last node时，它跟我们第一次读的时候version相同，
            //所以，把那个记录依旧留在rdset中，tx_read就会自动检查它,
            //在这里我们只重置另一个记录
            wrtset_s.size = 0;
            searched_value_s = lastNodeId;
            __syncwarp();
            tmpCurNodeId = -1;
            backFlag = true;
            //从这里出来的，searched_value_s所保留的nodeId一定不是leaf的.
            //所以这个循环算上这次至少要跑2次.
            //那么错误的lastNodeId最多存活在下一个循环里。（再下一个就被覆盖掉了.）
BACK: 
/*}}}*/
#endif
            lastNodeId = tmpCurNodeId;
            last_node_info = nodeInfo; 
            last_key = key;
            last_val = val;
            last_target_thread = target_thread_s;
           
            tmpCurNodeId = searched_value_s;

#ifdef ABORT_COUNT 
            if (tx_thread_id == 0)
                g_data.g_roll_back_count[mission_id].traversal_steps++; 
#endif 
            // read node
            if (read_node(tmpCurNodeId, key, val, nodeInfo)==false) { 
                goto TX_START1;
            }
            
            __syncwarp();
            
 //=========提前分裂 situation one 提前分裂满的internal node
/*{{{*/
            if ((getNodeSize(nodeInfo)==ORDER-1)&&(!isLeaf(nodeInfo))) {
                
                if (backFlag) goto TX_START;
                int up_key = -1;
                //Internal node split 提前分裂
                if (node_splitting(tmpCurNodeId, key, val, nodeInfo, up_key)==false) goto TX_START1;
                char place_for_new_loc;
                if (lastNodeId == -1){
                    if (insert_new_root(up_key, tmpCurNodeId, getNodeLevel(nodeInfo)+1, place_for_new_loc)==false) goto TX_START;
                    if (tx_commit(true, place_for_new_loc)==false) goto TX_START;
                }
                else {
                    if (insert_node(lastNodeId, up_key, last_key, last_val, last_target_thread, last_node_info, place_for_new_loc)==false) goto TX_START1;
                    if (tx_commit(false, place_for_new_loc)==false) goto TX_START1;
                }
                //tx_commit
                goto TX_START;
            }
            __syncwarp();
/*}}}*/
            //get target 
            get_target(key, val, target_key, searched_value_s, target_thread_s ); 
            
            __syncwarp();
            
            backFlag = false;
            
               
            /*}}}*/
        }while (!isLeaf(nodeInfo)); 
       //==================================SEARCH DONE==================================== 

S_WRITE:

#ifdef ABORT_COUNT
        __syncwarp();
        if (tx_thread_id == 0)
            g_data.g_roll_back_count[mission_id].roll_back_times = timesss;
        __syncwarp();
#endif
        __syncwarp();
     

        *ans = -1;
     
        __syncwarp();
        //判断是否找到结果。
        //如果能找到结果，结果必然是在target_thread_s-1的位置。
        if (key == target_key) { 
            *ans = val;
        }
        //有结果的赋值为结果，没有结果的一律-1
       
        //test
       
        __syncwarp();
       
        if (mission == GET) {
            continue;
        }

        __syncwarp();
        
        int up_key = -1; 

        if (insert_leaf(tmpCurNodeId, nodeInfo, target_key, target_val, key, val, target_thread_s, up_key) == false) {
            goto TX_START1;
        }
        if (up_key!=-1) {
            /*if (backFlag)  {
                printf("noway!\n");
                goto TX_START;
            }*/
            char place_for_new_loc;
            if (lastNodeId == -1) {
                if (insert_new_root(up_key, tmpCurNodeId,getNodeLevel(nodeInfo)+1, place_for_new_loc) == false) goto TX_START;
                if (tx_commit(true, place_for_new_loc)==false) goto TX_START;
            }
            else {
                if (insert_node(lastNodeId, up_key, last_key, last_val, last_target_thread, last_node_info, place_for_new_loc)==false) goto TX_START1; 
                if (tx_commit(false, place_for_new_loc)==false) goto TX_START1;
            }
        }
        else {
            if (tx_commit(false, NULL)==false) goto TX_START1;
        }
        
        //if (tx_thread_id==0) printf("success return %d\n", target_key);
        __syncwarp();
        continue; 





    }
    
 
    

    return;

}





__global__ void search_kernel( g_tree_t g_tree,
        g_data_t g_data,
        int boundary,       //从第boundary个tx开始
        int size           //算上boundary大小的size (这个size是以mission个数为单位.) 
        ){
    /*{{{*/
    const int tx_id = (blockDim.x * blockIdx.x + threadIdx.x) / WARPSIZE + boundary;
    
    //const int tx_thread_id = (gridDim.x * blockDim.x + threadIdx.x) % ORDER;
    const int tx_id_in_block = threadIdx.x / WARPSIZE;
    
    if (tx_id * tx_size_get >= size) return; //把一整个warp都在size外面的扔掉.
    
    const int id_in_ntg_get = threadIdx.x % ntg_size_get;
    const int ntg_id_in_block_get = threadIdx.x / ntg_size_get;
    const int ntg_id_in_tx_get = threadIdx.x % WARPSIZE / ntg_size_get;
    const int mission_id_get = tx_id * tx_size_get + ntg_id_in_tx_get;  
    const int mask_get = __ballot_sync(0xffffffff, mission_id_get<size);

   
    
 

    //mission_t mission_get;
    key_t target_key_get;
    ans_t *ans_get;
    if (mission_id_get >= size) return;
    
#ifdef ABORT_COUNT
    g_data.g_roll_back_count[mission_id_get].roll_back_times = 0;
    g_data.g_roll_back_count[mission_id_get].traversal_steps = 0;
    g_data.g_roll_back_count[mission_id_get].leaf_traversal_steps = 0;
    __syncwarp();
#endif        
    
    //mission_get = g_data.g_mission[mission_id_get]; 
    int idx = g_data.g_idx[mission_id_get];
    target_key_get = g_data.g_keys[idx];
    ans_get = &(g_data.g_ans[idx]);   
    
      
    offset_t local_root = *(g_tree.g_root);
    if (local_root == -1) {
        *ans_get = -1;
        return ;
    }    
    
    radical_search(target_key_get, *ans_get, local_root); 
    
    return;

/*}}}*/

}









void GPU_STM_Tree::launchKernel(int size, bool special, Time_Measure &t) {

    int grid_dim1 = (h_boundary * WARPSIZE + (Block_Dim_Put-1)) / Block_Dim_Put;
    int grid_dim2 = (h_device_working_num2 - h_boundary * WARPSIZE + (Block_Dim_Get-1)) / Block_Dim_Get;
    
   
    //cout<<"true working:\t"<<h_device_working_num2<<endl;
    //cout<<"grid_dim1: "<<grid_dim1<<endl;
    //cout<<"grid_dim2: "<<grid_dim2<<endl;
    //cout<<"working size:      "<<h_device_working_num2<<endl;
    //cout<<"boundary:          "<<h_boundary*tx_size_get<<endl;
    t.gpuTimeStart();
    if (grid_dim1 != 0) 
        tx_insert_kernel<<<grid_dim1, Block_Dim_Put>>>(g_tree, g_data, g_tx, h_boundary, h_device_working_num2);
    t.gpuTimeEnd();
    t.gpuTimeStart();
    if (grid_dim2 !=0) 
        search_kernel<<<grid_dim2, Block_Dim_Get>>> (g_tree, g_data, h_boundary, h_device_working_num2);
    t.gpuTimeEnd();
}


void GPU_STM_Tree::launchKernelBefore(int size,Time_Measure &t) {
    t.gpuTimeStart();
    
    //hipcub::DeviceRadixSort::SortPairs<key_t, int>(g_data_phase_1.d_temp_storage, g_data_phase_1.temp_storage_bytes, g_data_phase_1.g_keys, g_data_phase_2.g_keys, g_data_phase_1.g_idx, g_data_phase_2.g_idx, batch_size, 0,32, 0, true);
    hipcub::DeviceRadixSort::SortPairs<key_t, int>(g_data_phase_1.d_temp_storage, g_data_phase_1.temp_storage_bytes, g_data_phase_1.g_keys, g_data_phase_2.g_keys, g_data_phase_1.g_idx, g_data_phase_2.g_idx, size);
    t.gpuTimeEnd();
    t.gpuTimeStart();
    hipcub::DeviceRunLengthEncode::Encode(g_data_phase_2.d_temp_storage, g_data_phase_2.temp_storage_bytes, g_data_phase_2.g_keys, g_data_phase_3.g_keys, g_data_phase_3.g_count, g_data_phase_3.g_num, size);
    
    t.gpuTimeEnd();
    t.gpuTimeStart();
    //scan真正需要的item_num应该是encode得到的g_num,所以要把g_num传回来，然后就能把g_num的值作为scan的参数传入了.
    CUDA_ERROR_HANDLER(hipMemcpy(&h_device_working_num2, g_data_phase_3.g_num, sizeof(int), hipMemcpyDeviceToHost));
    
    hipcub::DeviceScan::ExclusiveSum(g_data_phase_3.d_temp_storage, g_data_phase_3.temp_storage_bytes, g_data_phase_3.g_count, g_data_phase_3.g_idx, h_device_working_num2);
    CUDA_ERROR_HANDLER(hipDeviceSynchronize());

    t.gpuTimeEnd();
    t.gpuTimeStart();
    //int tmpBlockDim = (Block_Dim_Get*4<=512)?Block_Dim_Get*4:512 ;   //4是随便写的
    int tmpBlockDim = 64;
    dim3 grid_dim( (size + tmpBlockDim-1) / tmpBlockDim );
    dim3 block_dim(tmpBlockDim);
    mergeMission<<<grid_dim, block_dim>>>(g_data_phase_2, g_data_phase_3, g_data_phase_4, g_data);
    t.gpuTimeEnd();

    t.gpuTimeStart();
    hipcub::DeviceRadixSort::SortPairs<short, int>(g_data_phase_4.d_temp_storage, g_data_phase_4.temp_storage_bytes, g_data_phase_4.g_mission, g_data.g_mission, g_data_phase_4.g_idx, g_data.g_idx, h_device_working_num2, 0, 2);
    
    t.gpuTimeEnd();
    t.gpuTimeStart();
    //CUDA_ERROR_HANDLER(hipDeviceSynchronize());
   
    dim3 grid_dim2( (h_device_working_num2 + tmpBlockDim-1) / tmpBlockDim );
    getBoundary0<<<1, 1>>>(g_data.g_boundary);
    getBoundary1<<<grid_dim2, block_dim>>>(g_data.g_mission, g_data.g_boundary, h_device_working_num2);
    getBoundary2<<<1, 1>>>(g_data.g_mission, g_data.g_boundary, h_device_working_num2);
    
    CUDA_ERROR_HANDLER(hipMemcpy( &h_boundary, g_data.g_boundary, sizeof(int), hipMemcpyDeviceToHost ));
    CUDA_ERROR_HANDLER(hipDeviceSynchronize());
    t.gpuTimeEnd();
}

void GPU_STM_Tree::launchKernelAfter(int size){
    dim3 grid_dim(Grid_Dim);
    dim3 block_dim(Block_Dim_Get / WARPSIZE * tx_size_get);
    writebackAns<<<grid_dim, block_dim>>>(g_data_phase_3, g_data, size);
}

__global__ void preprocessKernel(g_tree_t g_tree, g_tx_t g_tx){
    
    *(g_tree.g_root) = -1;
    *(g_tree.g_tree_size) = 0;

    g_tx.g_lock_table[MAX_NODE_NUM-1] = 0;    //root lock 初始化

}
__global__ void preprocessKernel_idx_init(int *g_idx_bef) {
    g_idx_bef[blockDim.x * blockIdx.x + threadIdx.x] = blockDim.x * blockIdx.x + threadIdx.x;
}




void GPU_STM_Tree::preprocess() {
    dim3 grid_dim(1);
    dim3 block_dim(1);
    preprocessKernel<<<grid_dim, block_dim>>>(g_tree, g_tx);
    dim3 grid_dim2(Grid_Dim);
    dim3 block_dim2(Block_Dim_Get / WARPSIZE * tx_size_get);
    preprocessKernel_idx_init<<<grid_dim2, block_dim2>>>(g_data_phase_1.g_idx);
    
    //为了得到temp_storage_bytes大小
    hipcub::DeviceRadixSort::SortPairs<key_t, int>(g_data_phase_1.d_temp_storage, g_data_phase_1.temp_storage_bytes, g_data_phase_1.g_keys, g_data_phase_2.g_keys, g_data_phase_1.g_idx, g_data_phase_2.g_idx, batch_size);
    CUDA_ERROR_HANDLER(hipMalloc(&(g_data_phase_1.d_temp_storage), g_data_phase_1.temp_storage_bytes ));

    hipcub::DeviceRunLengthEncode::Encode(g_data_phase_2.d_temp_storage, g_data_phase_2.temp_storage_bytes, g_data_phase_2.g_keys, g_data_phase_3.g_keys, g_data_phase_3.g_count, g_data_phase_3.g_num, batch_size);
    CUDA_ERROR_HANDLER(hipMalloc(&(g_data_phase_2.d_temp_storage), g_data_phase_2.temp_storage_bytes ));

    hipcub::DeviceScan::ExclusiveSum(g_data_phase_3.d_temp_storage, g_data_phase_3.temp_storage_bytes, g_data_phase_3.g_count, g_data_phase_3.g_idx, batch_size);
    CUDA_ERROR_HANDLER(hipMalloc(&(g_data_phase_3.d_temp_storage), g_data_phase_3.temp_storage_bytes ));

    hipcub::DeviceRadixSort::SortPairs<short, int>(g_data_phase_4.d_temp_storage, g_data_phase_4.temp_storage_bytes, g_data_phase_4.g_mission, g_data.g_mission, g_data_phase_4.g_idx, g_data.g_idx, batch_size, 0, 2);
    CUDA_ERROR_HANDLER(hipMalloc(&(g_data_phase_4.d_temp_storage), g_data_phase_4.temp_storage_bytes ));
    //for test
#if 0 
    cout<<"g_tree: "<<endl;
    cout<<(offset_t*)g_tree.g_root<<endl;
    cout<<(offset_t*)g_tree.g_tree_size<<endl;
    cout<<(keyArr_t*)g_tree.g_key_section<<endl;
    cout<<(valArr_t*)g_tree.g_val_section<<endl;
    cout<<"g_data: "<<endl;
    cout<<g_data.g_keys<<endl;
    cout<<g_data.g_vals<<endl;
    cout<<g_data.g_mission<<endl;
    cout<<g_data.g_ans<<endl;
    cout<<"g_tx: "<<endl;
    cout<<g_tx.g_wrtSet_location<<endl;
    cout<<g_tx.g_wrtSet_version<<endl;
    cout<<g_tx.g_wrtSet_content_key<<endl;
    cout<<g_tx.g_wrtSet_content_val<<endl;
    cout<<(version_t*)g_tx.g_lock_table<<endl;
#endif


}


















}
